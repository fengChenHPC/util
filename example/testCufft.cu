#include "../inc/yyfnutil.h"

#define NX 20
#define BATCH 1

#define CUDA_SAFE_CALL 
#define warn printf

void runtest1d() {
	float *h_idata = (float*) Malloc(NX*BATCH*sizeof(float));

	for(int i=0;i<NX*BATCH;i++)
	{
		h_idata[i]=i;
		warn("h_idata[%d].r=%f\n",i,h_idata[i]);
	}


 	hipfftHandle plan;
	hipfftComplex *d_data;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_data,sizeof(hipfftComplex)*NX*BATCH));
	CUDA_SAFE_CALL(hipMemcpy((hipfftReal*)d_data,h_idata,sizeof(float)*NX*BATCH,hipMemcpyHostToDevice));

	//Real to Complex
	hipfftPlan1d(&plan,NX,HIPFFT_R2C,BATCH);
	hipfftExecR2C(plan,(hipfftReal*)d_data,d_data);
	//CUDA_SAFE_CALL(hipMemcpy(h_odata,d_data,sizeof(hipfftComplex)*NX*BATCH,hipMemcpyDeviceToHost));
  	hipfftDestroy(plan);

	//Complex to Real
	hipfftPlan1d(&plan,NX,HIPFFT_C2R,BATCH);
	hipfftExecC2R(plan,d_data,(hipfftReal*)d_data);
	CUDA_SAFE_CALL(hipMemcpy(h_idata,d_data,sizeof(float)*NX*BATCH,hipMemcpyDeviceToHost));
printf("...............................................................................................\n");
	for(int i=0;i<NX*BATCH;i++)
	{
		//  warn("h_odata[%d].r=%f,h_odata[%d].i=%f",i,h_odata[i].r,i,h_odata[i].i);
		warn("h_idata[%d].r=%f\n",i,h_idata[i]/NX);
	}

	free(h_idata);
								 
/*	
	//need to divide the result by nx
	int nx = 20;
	Cufft::FFT1D *fft = new Cufft::FFT1D(nx, HIPFFT_C2C);
	hipfftComplex *data;
	size_t len = nx * sizeof(hipfftComplex);
	CudaMalloc((void**) &data, len);
	hipfftComplex *h_data = (hipfftComplex*) Malloc(nx * sizeof(hipfftComplex));
	for (int i = 0; i < nx; i++) {
		h_data[i].x = i;
		h_data[i].y = i;
	}
	CudaMemcpy(data, h_data,len, hipMemcpyHostToDevice);

	fft->c2c(data, data, HIPFFT_FORWARD);
	CudaMemcpy(h_data, data, len, hipMemcpyDeviceToHost);
	for (int i = 0; i < nx; i++) {
		printf("%d, %0.2f, %0.2f\n", i, h_data[i].x, h_data[i].y);
		h_data[i].x /= nx;
		h_data[i].y /= nx;
	}
	CudaMemcpy(data, h_data, len, cudaMemcpHostToDevice);
	
	fft->c2c(data, data, HIPFFT_BACKWARD);
	CudaMemcpy(h_data, data, len, hipMemcpyDeviceToHost);
	for (int i = 0; i < nx; i++) {
		printf("%d, %0.2f, %0.2f\n", i, h_data[i].x, h_data[i].y);
	}
	
	delete fft;
*/
}
int main() {
	runtest1d();
	return 0;
}
