#include <stdio.h>
#include <hipblas.h>

#include "../inc/yyfnutil.h"

int main(){
	hipblasHandle_t handle;
	CublasCreate(&handle);

	int n = 8;
	int k = n-1;

	float *A = (float*)Malloc(n*n*sizeof(float));
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n-i; j++){
			A[i*n+j] = 1.0f;
		}
	}

	float *x = (float*)Malloc(n*sizeof(float));
	for(int i = 0; i < n; i++){
		//x[i] = n-i;
		x[i] = i+1;
	}

	float *d_A;
	CudaMalloc((void**)&d_A, n*n*sizeof(float));
	CudaMemcpy(d_A, A, n*n*sizeof(float), hipMemcpyHostToDevice);

	float *d_x;
	CudaMalloc((void**)&d_x, n*sizeof(float));
	CudaMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	
	hipblasStbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, d_A, n, d_x, 1);

	CudaMemcpy(x, d_x, n*sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < n; i++){
		printf("%E\n", x[i]);
	}

	CublasDestroy(handle);

	CudaFree(d_x);
	CudaFree(d_A);
	free(A);
	free(x);

	return 0;
}
